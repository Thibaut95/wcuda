#include "hip/hip_runtime.h"
#include "Mandelbrot.h"

#include <iostream>
#include "Device.h"
#include <assert.h>
#include "MandelbrotMath.h"
#include <DomaineMath_GPU.h>
#include <Interval_GPU.h>

using std::cout;
using std::endl;

/*----------------------------------------------------------------------*\
 |*			Declaration 					*|
 \*---------------------------------------------------------------------*/
/*--------------------------------------*\
 |*		Imported	 	*|
 \*-------------------------------------*/

extern __global__ void mandelbrot(uchar4* ptrDevPixels, uint w, uint h, int t, DomaineMath domaineMath);

/*--------------------------------------*\
 |*		Public			*|
 \*-------------------------------------*/

/*--------------------------------------*\
 |*		Private			*|
 \*-------------------------------------*/

/*----------------------------------------------------------------------*\
 |*			Implementation 					*|
 \*---------------------------------------------------------------------*/

/*--------------------------------------*\
 |*		Public			*|
 \*-------------------------------------*/

Mandelbrot::Mandelbrot(const Grid& grid, uint w, uint h, int dt, const DomaineMath& domaineMath) :
	Animable_I<uchar4>(grid, w, h, "MandelBrot_CUDA_rgba_uchar4", domaineMath), variateurAnimation(Interval<uint>(20, 120), dt)
    {
    // Input
    this->dt = dt;

    // Tools
    this->t = 0;					// protected dans super classe Animable

    }

Mandelbrot::~Mandelbrot(void)
    {
    // rien
    }

/*--------------------------------------*\
 |*		Public			*|
 \*-------------------------------------*/

/**
 * Override
 */
void Mandelbrot::animationStep()
    {
    this->t = variateurAnimation.varierAndGet(); // in [0,2pi]
    }

/*--------------------------------------*\
 |*		Private			*|
 \*-------------------------------------*/

/**
 * Override (code entrainement cuda)
 */
void Mandelbrot::process(uchar4* ptrDevPixels, uint w, uint h, const DomaineMath& domaineMath)
    {
    Device::lastCudaError("mandelbrot rgba uchar4 (before kernel)"); // facultatif, for debug only, remove for release

    // le kernel est importer ci-dessus (ligne 19)
    float t = variateurAnimation.get();
    mandelbrot<<<dg,db>>>(ptrDevPixels,w,h,t, domaineMath);

    Device::lastCudaError("mandelbrot rgba uchar4 (after kernel)"); // facultatif, for debug only, remove for release
    }

/*--------------------------------------*\
 |*		Private			*|
 \*-------------------------------------*/

/*----------------------------------------------------------------------*\
 |*			End	 					*|
 \*---------------------------------------------------------------------*/

