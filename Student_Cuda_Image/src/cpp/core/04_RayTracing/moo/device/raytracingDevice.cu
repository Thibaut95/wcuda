#include "hip/hip_runtime.h"
#include "Indice2D.h"
#include "cudaTools.h"
#include "Device.h"

#include "IndiceTools_GPU.h"
#include "Sphere.h"
#include "RaytracingMath.h"
using namespace gpu;

// Attention : 	Choix du nom est impotant!
//		VagueDevice.cu et non Vague.cu
// 		Dans ce dernier cas, probl�me de linkage, car le nom du .cu est le meme que le nom d'un .cpp (host)
//		On a donc ajouter Device (ou n'importequoi) pour que les noms soient diff�rents!

/*----------------------------------------------------------------------*\
 |*			Declaration 					*|
 \*---------------------------------------------------------------------*/

/*--------------------------------------*\
 |*		Imported	 	*|
 \*-------------------------------------*/

/*--------------------------------------*\
 |*		Public			*|
 \*-------------------------------------*/

__global__ void raytracing(uchar4* ptrPixel, Sphere* ptrDevTabSphere, int nbSphere, uint w, uint h, float t);

/*--------------------------------------*\
 |*		Private			*|
 \*-------------------------------------*/

/*----------------------------------------------------------------------*\
 |*			Implementation 					*|
 \*---------------------------------------------------------------------*/

/*--------------------------------------*\
 |*		Public			*|
 \*-------------------------------------*/

__global__ void raytracing(uchar4* ptrPixel, Sphere* ptrDevTabSphere, int nbSphere, uint w, uint h, float t)
    {
    RaytracingMath raytracingMath = RaytracingMath(w, h, nbSphere, ptrDevTabSphere);

    const int TID = Indice2D::tid();
    const int NB_THREAD = Indice2D::nbThread();
    const int WH = w * h;
    int s = TID;

    int j; // in [0,w[
    int i; // in [0,h[

    while (s < WH)
	{
	IndiceTools::toIJ(s, w, &i, &j); // s[0,W*H[ --> i[0,H[ j[0,W[

	raytracingMath.colorIJ(&ptrPixel[s], i, j, t); // update ptrTabSpheres[s]

	s += NB_THREAD;
	}

    }

/*--------------------------------------*\
 |*		Private			*|
 \*-------------------------------------*/

/*----------------------------------------------------------------------*\
 |*			End	 					*|
 \*---------------------------------------------------------------------*/

