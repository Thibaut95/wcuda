#include "hip/hip_runtime.h"
#include <iostream>
#include <assert.h>

#include "Device.h"
#include <assert.h>
#include "Raytracing.h"
#include "SphereCreator.h"
#include "Sphere.h"

using std::cout;
using std::endl;

/*----------------------------------------------------------------------*\
 |*			Declaration 					*|
 \*---------------------------------------------------------------------*/

/*--------------------------------------*\
 |*		Imported	 	*|
 \*-------------------------------------*/

extern __global__ void raytracing(uchar4* ptrPixel, Sphere* ptrDevTabSphere, int nbSphere, uint w, uint h,float t);

/*--------------------------------------*\
 |*		Public			*|
 \*-------------------------------------*/

/*--------------------------------------*\
 |*		Private			*|
 \*-------------------------------------*/

/*----------------------------------------------------------------------*\
 |*			Implementation 					*|
 \*---------------------------------------------------------------------*/

/*--------------------------------------*\
 |*		Public			*|
 \*-------------------------------------*/

/*-------------------------*\
 |*	Constructeur	    *|
 \*-------------------------*/

Raytracing::Raytracing(int nbSphere, const Grid& grid, uint w, uint h, float dt) :
	Animable_I<uchar4>(grid, w, h, "Raytracing_Cuda_RGBA_uchar4")
    {
    assert(w == h); // specific rippling

    // Inputs
    this->dt = dt;

    // Tools
    this->t = 0; // protected dans Animable
    this->nbSphere = nbSphere;
    this->sizeOctetSpheres = nbSphere * sizeof(Sphere); //octets

    SphereCreator sphereCreator(nbSphere, w, h); //sur la pile

    Sphere* ptrTabSphere = sphereCreator.getTabSphere();

    toGM(ptrTabSphere);

    //toCM(ptrTabSphere); //a implémenter plus tard dans le TP Raytracing advanced

    }

Raytracing::~Raytracing()
    {
    //rien
    }

/*-------------------------*\
 |*	Methode		    *|
 \*-------------------------*/

/**
 * Override
 * Call periodicly by the API
 *
 * Note : domaineMath pas use car pas zoomable
 */
void Raytracing::process(uchar4* ptrPixel, uint w, uint h, const DomaineMath& domaineMath)
    {
    Device::lastCudaError("raytracing rgba uchar4 (before kernel)"); // facultatif, for debug only, remove for release

    // le kernel est importer ci-dessus (ligne 19)
    raytracing<<<dg,db>>>(ptrPixel, ptrDevTabSphere, nbSphere ,w,h,t);

    Device::lastCudaError("raytracing rgba uchar4 (after kernel)"); // facultatif, for debug only, remove for release

    }

/**
 * Override
 * Call periodicly by the API
 */
void Raytracing::animationStep()
    {
    t += dt;
    }

/*--------------------------------------*\
 |*		Private			*|
 \*-------------------------------------*/
/**
 * Copy the host tab sphere to the device
 */
void Raytracing::toGM(Sphere* ptrDevSphere)
    {
     Device::memcpyHToD(ptrDevTabSphere, ptrDevSphere, sizeOctetSpheres);
    }

/*----------------------------------------------------------------------*\
 |*			End	 					*|
 \*---------------------------------------------------------------------*/

